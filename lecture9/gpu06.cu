
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void vec_add(double* c, const double* a, const double* b, long N){
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    c[i] = a[i] + b[i];
  }
}

__global__
void vec_add_kernel(double* c, const double* a, const double* b, long N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) c[idx] = a[idx] + b[idx];
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

int main() {
  long N = (1UL<<25);

  double *x, *y, *z;
  hipMallocManaged(&x, N * sizeof(double));
  hipMallocManaged(&y, N * sizeof(double));
  hipMallocManaged(&z, N * sizeof(double));
  double* z_ref = (double*) malloc(N * sizeof(double));
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    x[i] = i+2;
    y[i] = 1.0/(i+1);
    z[i] = 0;
    z_ref[i] = 0;
  }

  double tt = omp_get_wtime();
  vec_add(z_ref, x, y, N);
  printf("CPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  tt = omp_get_wtime();
  vec_add_kernel<<<N/1024,1024>>>(z, x, y, N);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double err = 0;
  for (long i = 0; i < N; i++) err += fabs(z[i]-z_ref[i]);
  printf("Error = %f\n", err);

  return 0;
}
