/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
// https://devblogs.nvidia.com/how-access-global-memory-efficiently-cuda-c-kernels/

#include <hip/hip_runtime.h>
#include <stdio.h>
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %sn", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void stride(double* a, int s)
{
  int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
  a[i] = a[i] + 1;
}

int main(int argc, char**argv)
{
  int blockSize = 256;
  int nMB = 4;
  float ms;

  double *d_a;
  hipEvent_t startEvent, stopEvent;

  int n = nMB*1024*1024/sizeof(double);

  // NB:  d_a(33*nMB) for stride case
  checkCuda( hipMalloc(&d_a, n * 33 * sizeof(double)) );

  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  printf("Stride, Bandwidth (GB/s):\n");

  for (int i = 1; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0.0, n * sizeof(double)) );

    checkCuda( hipEventRecord(startEvent,0) );
    stride<<<n/blockSize, blockSize>>>(d_a, i);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nMB/ms);
  }

  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  hipFree(d_a);
  return 0;
}
