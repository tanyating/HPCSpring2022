
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void vec_add(double* c, const double* a, const double* b, long N){
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    c[i] = a[i] + b[i];
  }
}

__global__
void vec_add_kernel(double* c, const double* a, const double* b, long N, long offset){
  int idx = offset + blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) c[idx] = a[idx] + b[idx];
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

int main() {
  //long N = (1UL<<25);

  const int blockSize = 1024, nStreams = 4;
  long N = 10000 * blockSize * nStreams;
  const int streamSize = N / nStreams;
  const int streamBytes = streamSize * sizeof(double);

  printf("N: %ld\n", N);
  printf("streamSize: %d\n", streamSize);

  double *x, *y, *z;
  hipHostMalloc((void**)&x, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&y, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&z, N * sizeof(double), hipHostMallocDefault);
  double* z_ref = (double*) malloc(N * sizeof(double));
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    x[i] = i+2;
    y[i] = 1.0/(i+1);
    z[i] = 0;
    z_ref[i] = 0;
  }

  double tt = omp_get_wtime();
  vec_add(z_ref, x, y, N);
  printf("CPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double *x_d, *y_d, *z_d;
  hipMalloc(&x_d, N*sizeof(double));
  hipMalloc(&y_d, N*sizeof(double));
  hipMalloc(&z_d, N*sizeof(double));

  tt = omp_get_wtime();
  hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(y_d, y, N*sizeof(double), hipMemcpyHostToDevice);
  vec_add_kernel<<<N/1024,1024>>>(z_d, x_d, y_d, N, 0);
  hipMemcpyAsync(z, z_d, N*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double err = 0;
  for (long i = 0; i < N; i++) err += fabs(z[i]-z_ref[i]);
  printf("Error = %f\n", err);

  hipStream_t stream[nStreams];
  for (int i = 0; i < nStreams; ++i)
    hipStreamCreate(&stream[i]);

  tt = omp_get_wtime();
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    hipMemcpyAsync(&x_d[offset], &x[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]);
    hipMemcpyAsync(&y_d[offset], &y[offset],
                               streamBytes, hipMemcpyHostToDevice,
                               stream[i]);
    vec_add_kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(z_d, x_d, y_d, N, offset);
    hipMemcpyAsync(&z[offset], &z_d[offset],
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]);
  }
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  err = 0;
  for (long i = 0; i < N; i++) err += fabs(z[i]-z_ref[i]);
  printf("Error = %f\n", err);

  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);

  hipHostFree(x);
  hipHostFree(y);
  hipHostFree(z);
  free(z_ref);

  return 0;
}

